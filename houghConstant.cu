#include "hip/hip_runtime.h"
/*
 ============================================================================
 Transformada de Hough con Memoria CONSTANTE
 
 Para usar: Modificar Makefile a houghConstant.cu
 Para ejecutar: ./houghConstant <nombre_imagen>
 ============================================================================
 */
 
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "common/pgm.h"
#include "common/draw.h"

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;
//*****************************************************************

//*****************************************************************
// MEMORIA CONSTANTE
// Usar memoria constante para la tabla de senos y cosenos
// inicializarlas en main y pasarlas al device
__constant__ float d_Cos[degreeBins];
__constant__ float d_Sin[degreeBins];
//*****************************************************************

//**********************************************************
// Función para calcular el umbral

int calculateThreshold(int *accumulator, int size, float factor) {
    
    int maxVal = 0;
    for (int i = 0; i < size; i++) {
        if (accumulator[i] > maxVal) maxVal = accumulator[i];
    }
    
    return maxVal * factor;
}

//*********************************************************


// The CPU function returns a pointer to the accummulator
void CPU_HoughTran (unsigned char *pic, int w, int h, int **acc)
{
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;  //(w^2 + h^2)/2, radio max equivalente a centro -> esquina
  *acc = new int[rBins * degreeBins];            //el acumulador, conteo depixeles encontrados, 90*180/degInc = 9000
  memset (*acc, 0, sizeof (int) * rBins * degreeBins); //init en ceros
  int xCent = w / 2;
  int yCent = h / 2;
  float rScale = 2 * rMax / rBins;

  for (int i = 0; i < w; i++) //por cada pixel
    for (int j = 0; j < h; j++) //...
      {
        int idx = j * w + i;
        if (pic[idx] > 0) //si pasa thresh, entonces lo marca
          {
            int xCoord = i - xCent;
            int yCoord = yCent - j;  // y-coord has to be reversed
            float theta = 0;         // actual angle
            for (int tIdx = 0; tIdx < degreeBins; tIdx++) //add 1 to all lines in that pixel
              {
                float r = xCoord * cos (theta) + yCoord * sin (theta);
                int rIdx = (r + rMax) / rScale;
                (*acc)[rIdx * degreeBins + tIdx]++; //+1 para este radio r y este theta
                theta += radInc;
              }
          }
      }
}

//*****************************************************************
//TODO Kernel memoria compartida
// __global__ void GPU_HoughTranShared(...)
// {
//   //TODO
// }

// GPU kernel. One thread per image pixel is spawned.
// The accummulator memory needs to be allocated by the host in global memory
__global__ void GPU_HoughTranConst (unsigned char *pic, int w, int h, int *acc, float rMax, float rScale)
{
  // Calcular: int gloID
  int gloID = blockIdx.x * blockDim.x + threadIdx.x; 
  if (gloID > w * h) return;      // in case of extra threads in block

  int xCent = w / 2;
  int yCent = h / 2;

  //TODO explicar bien bien esta parte. Dibujar un rectangulo a modo de imagen sirve para visualizarlo mejor
  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  //TODO eventualmente usar memoria compartida para el acumulador

  if (pic[gloID] > 0)
    {
      for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
          //TODO utilizar memoria constante para senos y cosenos
          //float r = xCoord * cos(tIdx) + yCoord * sin(tIdx); //probar con esto para ver diferencia en tiempo
          float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
          int rIdx = (r + rMax) / rScale;
          //debemos usar atomic, pero que race condition hay si somos un thread por pixel? explique
          atomicAdd (acc + (rIdx * degreeBins + tIdx), 1);
        }
    }

  //TODO eventualmente cuando se tenga memoria compartida, copiar del local al global
  //utilizar operaciones atomicas para seguridad
  //faltara sincronizar los hilos del bloque en algunos lados

}



//*****************************************************************
int main (int argc, char **argv)
{
  int i;

  PGMImage inImg (argv[1]);
  
  // Leer imagen usando OpenCV desde draw.cpp
  cv::Mat inputImage = loadImage(argv[1]);
  if (inputImage.empty()) {
      printf("Error al cargar la imagen.\n");
      return -1;
  }

  cv::Mat processedImage;
  GaussianBlur(inputImage, inputImage, Size(3,3), 0);
  Canny(inputImage, processedImage, 30, 100);
  dilate(processedImage, processedImage, Mat(), Point(-1,-1), 1);

  unsigned char* pixels = processedImage.data;

  int *cpuht;
  int w = inputImage.cols;
  int h = inputImage.rows;

  //float* d_Cos;
  //float* d_Sin;

  // MEMORIA CONSTANTE
  // YA NO ES NECESARIO RESERVAR MEMORIA EN EL DEVICE PARA:
  //hipMalloc ((void **) &d_Cos, sizeof (float) * degreeBins);
  //hipMalloc ((void **) &d_Sin, sizeof (float) * degreeBins);

  // CPU calculation
  CPU_HoughTran(pixels, w, h, &cpuht);

  // pre-compute values to be stored
  float *pcCos = (float *) malloc (sizeof (float) * degreeBins);
  float *pcSin = (float *) malloc (sizeof (float) * degreeBins);
  float rad = 0;
  for (i = 0; i < degreeBins; i++)
  {
    pcCos[i] = cos (rad);
    pcSin[i] = sin (rad);
    rad += radInc;
  }

  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  float rScale = 2 * rMax / rBins;

  // MEMORIA CONSTANTE
  // Copiar los valores precalculados de seno y coseno a memoria constante
  hipMemcpyToSymbol(HIP_SYMBOL(d_Cos), pcCos, sizeof(float) * degreeBins);
  hipMemcpyToSymbol(HIP_SYMBOL(d_Sin), pcSin, sizeof(float) * degreeBins);

  // setup and copy data from host to device
  unsigned char *d_in, *h_in;
  int *d_hough, *h_hough;

  h_in = pixels; // h_in contiene los pixeles de la imagen

  h_hough = (int *) malloc (degreeBins * rBins * sizeof (int));

  hipMalloc ((void **) &d_in, sizeof (unsigned char) * w * h);
  hipMalloc ((void **) &d_hough, sizeof (int) * degreeBins * rBins);
  hipMemcpy (d_in, h_in, sizeof (unsigned char) * w * h, hipMemcpyHostToDevice);
  hipMemset (d_hough, 0, sizeof (int) * degreeBins * rBins);

    // Definir eventos de inicio y fin
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Inicia la medicion de tiempo
  hipEventRecord(start, 0);

  // execution configuration uses a 1-D grid of 1-D blocks, each made of 256 threads
  //1 thread por pixel

  // MEMORIA CONSTANTE:
  // Ya no es necesario pasar las referencias de d_Cos y d_Sin como parámetros al kernel
  int blockNum = ceil (w * h / 256);
  GPU_HoughTranConst <<< blockNum, 256 >>> (d_in, w, h, d_hough, rMax, rScale);

  // get results from device
  hipMemcpy (h_hough, d_hough, sizeof (int) * degreeBins * rBins, hipMemcpyDeviceToHost);

  // Termina la medicion de tiempo
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  // Calcula el tiempo en milisegundos
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("Tiempo de ejecución del kernel: %f ms\n", elapsedTime);

  // Liberar eventos
  hipEventDestroy(start);
  hipEventDestroy(stop);

  //*************************************************** */

  // Seccion de output

  // Calcular el umbral
  int threshold = calculateThreshold(h_hough, degreeBins * rBins, 0.46);

  int xCent = w / 2;
  int yCent = h / 2;

  printf("threshold: %d\n", threshold);

  // Dibujar líneas detectadas
  drawLines(inputImage, h_hough, threshold, rMax, rScale, degreeBins, radInc, rBins, xCent, yCent);

  // Guardar imagen con líneas detectadas
  saveImage("output_with_lines.png", inputImage);
  printf("Imagen de salida guardada como output_with_lines.png\n");

  // compare CPU and GPU results
  for (i = 0; i < degreeBins * rBins; i++)
  {
    if (cpuht[i] != h_hough[i])
      printf ("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
  }

  // ********************************************************************************

  free(cpuht);
  free(pcCos);
  free(pcSin);
  free(h_hough);
  hipFree(d_in);
  hipFree(d_hough);

  // MEMORIA CONSTANTE:
  // Ya no es necesario liberar d_Cos y d_Sin

  //hipFree(d_Cos);
  //hipFree(d_Sin);
  return 0;
}